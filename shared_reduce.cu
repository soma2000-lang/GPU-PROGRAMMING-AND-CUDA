#include <iostream>
#include <hip/hip_runtime.h>

__global__ void SimpleSumReductionKernel(float* input, float* output) {
    unsigned int i = 2 * threadIdx.x;
    for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2) {
        if (threadIdx.x % stride == 0) {
            input[i] += input[i + stride];
        }
        __syncthreads();

    }
    if (threadIdx.x == 0) {
    *output = input[0];
    }
}

int main() {
    // Size of the input data
    const int size = 2048;
    const int bytes = size * sizeof(float);

    // Allocate memory for input and output on host
    float* h_input = new float[size];
    float* h_output = new float;

    // Initialize input data on host
    for (int i = 0; i < size; i++) {
        h_input[i] = 1.0f; // Example: Initialize all elements to 1
    }

    // Allocate memory for input and output on device
    float* d_input;
    float* d_output;
    hipMalloc(&d_input, bytes);
    hipMalloc(&d_output, sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_input, h_input, bytes, hipMemcpyHostToDevice);

    // Launch the kernel
    SimpleSumReductionKernel<<<1, size / 2>>>(d_input, d_output);

    // Copy result back to host
    hipMemcpy(h_output, d_output, sizeof(float), hipMemcpyDeviceToHost);

    // Print the result
    std::cout << "Sum is " << *h_output << std::endl;

    // Cleanup
    delete[] h_input;
    delete h_output;
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}
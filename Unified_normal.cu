


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cassert>
#include <iostream>

using std::cout;


__global__ void vectorAdd(int *a, int *b, int *c, int N)
{

    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;

    if (tid < N)
    {
        c[tid] = a[tid] + b[tid];
    }
}

int main()
{

    const int N = 1 << 16;
    size_t bytes = N * sizeof(int);

    int *a, *b, *c;

    hipMallocManaged(&a, bytes);
    hipMallocManaged(&b, bytes);
    hipMallocManaged(&c, bytes);




    for (int i = 0; i < N; i++)
    {
        a[i] = rand() % 100;
        b[i] = rand() % 100;
    }

    int BLOCK_SIZE = 1 << 10;

    int GRID_SIZE = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    vectorAdd<<<GRID_SIZE, BLOCK_SIZE>>>(a, b, c, N);

    hipDeviceSynchronize();

    for (int i = 0; i < N; i++)
    {
        assert(c[i] == a[i] + b[i]);
    }


    hipFree(a);
    hipFree(b);
    hipFree(c);

    cout << "COMPLETED SUCCESSFULLY!\n";

    return 0;
}
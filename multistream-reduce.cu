#include <iostream>
#include <hip/hip_runtime.h>

#define BLOCK_DIM 1024
#define COARSE_FACTOR 2
#define NUM_DEVICES 2

__global__ void CoarsenedReduction(float* input, float* output, int size) {
    __shared__ float input_s[BLOCK_DIM];

    unsigned int i = blockIdx.x * blockDim.x * COARSE_FACTOR + threadIdx.x;
    unsigned int t = threadIdx.x;
    float sum = 0.0f;

    // Reduce within a thread
    for (unsigned int tile = 0; tile < COARSE_FACTOR; ++tile) {
        unsigned int index = i + tile * blockDim.x;
        if (index < size) {
            sum += input[index];
        }
    }

    input_s[t] = sum;
    __syncthreads();

    // Reduce within a block
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (t < stride) {
            input_s[t] += input_s[t + stride];
        }
        __syncthreads();
    }

    // Reduce over blocks
    if (t == 0) {
        atomicAdd(output, input_s[0]);
    }
}

int main() {
    const int size = 10000;
    const int bytes = size * sizeof(float);

    // Allocate memory for input and output on host
    float* h_input = new float[size];
    float* h_output = new float;

    // Initialize input data on host
    for (int i = 0; i < size; i++) {
        h_input[i] = 1.0f;  // Example: Initialize all elements to 1
    }

    // Create CUDA streams for pipelining
    hipStream_t streams[NUM_DEVICES];
    for (int i = 0; i < NUM_DEVICES; ++i) {
        hipSetDevice(i);
        hipStreamCreate(&streams[i]);
    }

    // Allocate memory for input and output on each device
    float* d_input[NUM_DEVICES];
    float* d_output[NUM_DEVICES];
    for (int i = 0; i < NUM_DEVICES; ++i) {
        hipSetDevice(i);
        hipMalloc(&d_input[i], bytes);
        hipMalloc(&d_output[i], sizeof(float));
        hipMemset(d_output[i], 0, sizeof(float));  // Initialize output to 0
    }

    // Copy data from host to each device
    for (int i = 0; i < NUM_DEVICES; ++i) {
        hipSetDevice(i);
        hipMemcpyAsync(d_input[i], h_input, bytes, hipMemcpyHostToDevice, streams[i]);
    }

    // Launch the kernel with coarsening on each device
    int numBlocks = (size + BLOCK_DIM * COARSE_FACTOR - 1) / (BLOCK_DIM * COARSE_FACTOR);
    for (int i = 0; i < NUM_DEVICES; ++i) {
        hipSetDevice(i);
        CoarsenedReduction<<<numBlocks, BLOCK_DIM, 0, streams[i]>>>(d_input[i], d_output[i], size);
    }

    // Copy results back to host from each device
    float* d_output_host[NUM_DEVICES];
    for (int i = 0; i < NUM_DEVICES; ++i) {
        hipHostMalloc(&d_output_host[i], sizeof(float), hipHostMallocDefault);
        hipSetDevice(i);
        hipMemcpyAsync(d_output_host[i], d_output[i], sizeof(float), hipMemcpyDeviceToHost, streams[i]);
    }

    // Wait for all streams to complete
    for (int i = 0; i < NUM_DEVICES; ++i) {
        hipSetDevice(i);
        hipStreamSynchronize(streams[i]);
    }

    // Sum the results from each device on the host
    float final_sum = 0.0f;
    for (int i = 0; i < NUM_DEVICES; ++i) {
        final_sum += *d_output_host[i];
    }

    // Print the result
    std::cout << "Sum is " << final_sum << std::endl;

    // Cleanup
    delete[] h_input;
    delete h_output;
    for (int i = 0; i < NUM_DEVICES; ++i) {
        hipSetDevice(i);
        hipFree(d_input[i]);
        hipFree(d_output[i]);
        hipHostFree(d_output_host[i]);
        hipStreamDestroy(streams[i]);
    }

    return 0;
}
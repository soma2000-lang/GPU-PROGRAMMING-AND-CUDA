#include <iostream>
#include <hip/hip_runtime.h>

#define BLOCK_DIM 1024

__global__ void SharedMemoryReduction(float* input, float* output, int n) {
    __shared__ float input_s[BLOCK_DIM]; 
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; // index within a block
    unsigned int t = threadIdx.x; // global index

    // Load elements into shared memory
    if (idx < n) {
        input_s[t] = input[idx];
    } else {
        input_s[t] = 0.0f;
    }
    __syncthreads();

    // Reduction in shared memory
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (t < stride && idx + stride < n) {
            input_s[t] += input_s[t + stride];
        }
        __syncthreads();
    }

    // Reduction across blocks in global memory
    // needs to be atomic to avoid contention
    if (t == 0) {
        atomicAdd(output, input_s[0]);
    }
}



int main() {
    // Size of the input data
    const int size = 100000;
    const int bytes = size * sizeof(float);

    // Allocate memory for input and output on host
    float* h_input = new float[size];
    float* h_output = new float;

    // Initialize input data on host
    for (int i = 0; i < size; i++) {
        h_input[i] = 1.0f; // Example: Initialize all elements to 1
    }

    // Allocate memory for input and output on device
    float* d_input;
    float* d_output;

    hipMalloc(&d_input, bytes);
    hipMalloc(&d_output, sizeof(float));

    // Copy data from host to device
    float zero = 0.0f;
    hipMemcpy(d_output, &zero, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_input, h_input, bytes, hipMemcpyHostToDevice);

    // Launch the kernel
    int numBlocks = (size + BLOCK_DIM - 1) / BLOCK_DIM;
    SharedMemoryReduction<<<numBlocks, BLOCK_DIM>>>(d_input, d_output, size);

    // Copy result back to host
    hipMemcpy(h_output, d_output, sizeof(float), hipMemcpyDeviceToHost);

    // Print the result
    std::cout << "Sum is " << *h_output << std::endl;

    // Cleanup
    delete[] h_input;
    delete h_output;
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}
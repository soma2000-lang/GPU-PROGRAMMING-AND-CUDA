
#include <hip/hip_runtime.h>
#include <stdio.h>

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
    if (result != cudaSuccess)
    {
        fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
        assert(result == cudaSuccess);
    }
#endif
    return result;
}

__global__ void kernel(float *a, int offset)
{
    int i = offset + threadIdx.x + blockIdx.x * blockDim.x;
    float x = (float)i;
    float s = sinf(x);
    float c = cosf(x);
    a[i] = a[i] + sqrtf(s * s + c * c);
}

float maxError(float *a, int n)
{
    float maxE = 0;
    for (int i = 0; i < n; i++)
    {
        float error = fabs(a[i] - 1.0f);
        if (error > maxE)
            maxE = error;
    }
    return maxE;
}

int main()
{
    const int blockSize = 768, nStreams = 4;
    const int n = 256 * 256;
    const int streamSize = n / nStreams;
    const int streamBytes = streamSize * sizeof(float);
    const int bytes = n * sizeof(float);
    float ms;

    // Host array
    float *a;
    hipHostMalloc((void **)&a, bytes, hipHostMallocDefault);
    memset(a, 0, bytes);

    // Device pointer array
    float **d_a = (float **)malloc(nStreams * sizeof(float *));
    hipStream_t stream[nStreams];

    hipEvent_t startEvent, stopEvent;
    checkCuda(hipEventCreate(&startEvent));
    checkCuda(hipEventCreate(&stopEvent));

    for (int i = 0; i < nStreams; ++i)
    {
        hipStreamCreate(&stream[i]);
        int offset = i * streamSize;

        checkCuda(hipMallocAsync((void **)&d_a[i], streamBytes, stream[i])); // Allocate memory on the device asynchronously

        checkCuda(hipMemcpyAsync(d_a[i], &a[offset], streamBytes, hipMemcpyHostToDevice, stream[i]));

        kernel<<<streamSize / blockSize, blockSize, 0, stream[i]>>>(*d_a, offset);

        checkCuda(hipMemcpyAsync(&a[offset], d_a[i], streamBytes, hipMemcpyDeviceToHost, stream[i]));
    }

    checkCuda(hipEventRecord(stopEvent, 0));
    checkCuda(hipEventSynchronize(stopEvent));
    checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
    printf("Time for sequential transfer and execute (ms): %f\n", ms);
    printf("  max error: %e\n", maxError(a, n));

    // Synchronize and clean up
    checkCuda(hipEventDestroy(startEvent));
    checkCuda(hipEventDestroy(stopEvent));
    for (int i = 0; i < nStreams; ++i)
        checkCuda(hipStreamDestroy(stream[i]));

    hipFree(d_a);
    hipHostFree(a);

    return 0;
}
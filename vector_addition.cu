#include <hip/hip_runtime.h>
#include <stdio.h>

// compute vector sum C = A + B
// each thread peforms one pair-wise addition
__global__ void vecAddKernel(float *A, float *B, float *C, int n) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < n) {
    C[i] = A[i] + B[i];
  }
}

// https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) {
      exit(code);
    }
  }
}

inline unsigned int cdiv(unsigned int a, unsigned int b) {
  return (a + b - 1) / b;
}

void vecAdd(float *A, float *B, float *C, int n) {
  float *A_d, *B_d, *C_d;
  size_t size = n * sizeof(float);

  hipMalloc((void **)&A_d, size);
  hipMalloc((void **)&B_d, size);
  hipMalloc((void **)&C_d, size);

  hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);
  hipMemcpy(B_d, B, size, hipMemcpyHostToDevice);

  const unsigned int numThreads = 256;
  unsigned int numBlocks = cdiv(n, numThreads);

  vecAddKernel<<<numBlocks, numThreads>>>(A_d, B_d, C_d, n);
  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());

  hipMemcpy(C, C_d, size, hipMemcpyDeviceToHost);

  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);
}

int main() {
  const int n = 1000;
  float A[n];
  float B[n];
  float C[n];

  // generate some dummy vectors to add
  for (int i = 0; i < n; i += 1) {
    A[i] = float(i);
    B[i] = A[i] / 1000.0f;
  }

  vecAdd(A, B, C, n);

  // print result
  for (int i = 0; i < n; i += 1) {
    if (i > 0) {
      printf(", ");
      if (i % 10 == 0) {
        printf("\n");
      }
    }
    printf("%8.3f", C[i]);
  }
  printf("\n");
  return 0;
}
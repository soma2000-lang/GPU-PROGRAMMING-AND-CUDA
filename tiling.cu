
#include <hip/hip_runtime.h>
__global__ void matrixMulSimple(float* A, float* B, float* C, int width) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if(col < width && row < width) {
        float value = 0;
        for (int k = 0; k < width; ++k) {
            value += A[row * width + k] * B[k * width + col];
        }
        C[row * width + col] = value;
    }
}

#define TILE_WIDTH 16

__global__ void matrixMulTiled(float* A, float* B, float* C, int width) {
    __shared__ float As[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Bs[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;
    int Row = by * blockDim.y + ty;
    int Col = bx * blockDim.x + tx;

    float value = 0;
    for (int m = 0; m < width/TILE_WIDTH; ++m) {
        As[ty][tx] = A[Row*width + (m*TILE_WIDTH + tx)];
        Bs[ty][tx] = B[(m*TILE_WIDTH + ty)*width + Col];
        __syncthreads();

        for (int k = 0; k < TILE_WIDTH; ++k) {
            value += As[ty][k] * Bs[k][tx];
        }
        __syncthreads();
    }

    if(Row < width && Col < width) {
        C[Row*width + Col] = value;
    }
}

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Size of the vector
#define N 100

// CUDA kernel to add two vectors
__global__ void vectorAdd(int *a, int *b, int *c) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        c[tid] = a[tid] + b[tid];
    }
}
int main() {
    int *h_a, *h_b, *h_c; // Host vectors
    int *d_a, *d_b, *d_c; // Device vectors

    // Initialize host vectors
    h_a = (int *)malloc(N * sizeof(int));
    h_b = (int *)malloc(N * sizeof(int));
    h_c = (int *)malloc(N * sizeof(int));

    // Initialize host vectors with random values
    for (int i = 0; i < N; i++) {
        h_a[i] = rand() % 10;
        h_b[i] = rand() % 10;
    }
    hipMalloc((void **)&d_a, N * sizeof(int));
    hipMalloc((void **)&d_b, N * sizeof(int));
    hipMalloc((void **)&d_c, N * sizeof(int));

    hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(int), hipMemcpyHostToDevice);
     vectorAdd<<<2, 50>>>(d_a, d_b, d_c);

    // Copy the result from GPU to CPU
    hipMemcpy(h_c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

    // Print the result
    for (int i = 0; i < N; i++) {
        printf("h_a[%d] %d + h_b[%d] %d = %d\n", i, h_a[i], i, h_b[i], h_c[i]);
    }
   for (int i = 0; i < N; i++) {
        printf("h_a[%d] %d + h_b[%d] %d = %d\n", i, h_a[i], i, h_b[i], h_c[i]);
    }

    // Free memory
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
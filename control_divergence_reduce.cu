#include <iostream>
#include <hip/hip_runtime.h>

__global__ void FixDivergenceKernel(float* input, float* output) {
    unsigned int i = threadIdx.x; //threads start next to each other
    for (unsigned int stride = blockDim.x; stride >= 1; stride /= 2) { // furthest element is blockDim away
        if (threadIdx.x < stride) { // 
            input[i] += input[i + stride]; // each thread adds a distant element to its assigned position
        }
        __syncthreads();

    }
    if (threadIdx.x == 0) {
    *output = input[0];
    }
}

int main() {
    // Size of the input data
    const int size = 2048;
    const int bytes = size * sizeof(float);

    // Allocate memory for input and output on host
    float* h_input = new float[size];
    float* h_output = new float;

    // Initialize input data on host
    for (int i = 0; i < size; i++) {
        h_input[i] = 1.0f; // Example: Initialize all elements to 1
    }

    // Allocate memory for input and output on device
    float* d_input;
    float* d_output;
    hipMalloc(&d_input, bytes);
    hipMalloc(&d_output, sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_input, h_input, bytes, hipMemcpyHostToDevice);

    // Launch the kernel
    FixDivergenceKernel<<<1, size / 2>>>(d_input, d_output);

    // Copy result back to host
    hipMemcpy(h_output, d_output, sizeof(float), hipMemcpyDeviceToHost);

    // Print the result
    std::cout << "Sum is " << *h_output << std::endl;

    // Cleanup
    delete[] h_input;
    delete h_output;
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}
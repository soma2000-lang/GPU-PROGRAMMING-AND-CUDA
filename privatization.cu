#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA kernel for vector addition without privatization
__global__ void vectorAdd(const float *a, const float *b, float *result, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        result[index] = a[index] + b[index];
    }
}

// CUDA kernel for vector addition with privatization
__global__ void vectorAddPrivatized(const float *a, const float *b, float *result, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        float a_private = a[index]; // Load into private memory
        float b_private = b[index]; // Load into private memory
        result[index] = a_private + b_private;
    }
}

// Function to initialize the vectors with dummy data
void initData(float *data, int size) {
    for (int i = 0; i < size; ++i) {
        data[i] = i;
    }
}

int main() {
    int n = 1<<20; // Size of the vectors
    float *a, *b, *result, *d_a, *d_b, *d_result;

    // Allocate memory on the host
    a = (float*)malloc(n * sizeof(float));
    b = (float*)malloc(n * sizeof(float));
    result = (float*)malloc(n * sizeof(float));

    // Initialize vectors
    initData(a, n);
    initData(b, n);

    // Allocate memory on the device
    hipMalloc(&d_a, n * sizeof(float));
    hipMalloc(&d_b, n * sizeof(float));
    hipMalloc(&d_result, n * sizeof(float));

    // Copy vectors from host to device
    hipMemcpy(d_a, a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(float), hipMemcpyHostToDevice);

    // Define number of blocks and threads
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the vector addition kernel without privatization
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_result, n);

    // Copy result back to host
    hipMemcpy(result, d_result, n * sizeof(float), hipMemcpyDeviceToHost);

    // Launch the vector addition kernel with privatization
    vectorAddPrivatized<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_result, n);

    // Copy result back to host
    hipMemcpy(result, d_result, n * sizeof(float), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);
    free(a);
    free(b);
    free(result);

    return 0;
}
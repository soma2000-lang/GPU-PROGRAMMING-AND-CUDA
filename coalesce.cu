#include <iostream>
#include <hip/hip_runtime.h>

__global__ void copyDataNonCoalesced(float *in, float *out, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        out[index] = in[(index * 2) % n];
    }
}

__global__ void copyDataCoalesced(float *in, float *out, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        out[index] = in[index];
    }
}

void initializeArray(float *arr, int n) {
    for(int i = 0; i < n; ++i) {
        arr[i] = static_cast<float>(i);
    }
}

int main() {
    const int n = 1 << 24; // Increase n to have a larger workload
    float *in, *out;

    hipMallocManaged(&in, n * sizeof(float));
    hipMallocManaged(&out, n * sizeof(float));

    initializeArray(in, n);

    int blockSize = 128; // Define block size
    // int blockSize = 1024; // change this when talking about occupancy
    int numBlocks = (n + blockSize - 1) / blockSize; // Ensure there are enough blocks to cover all elements

    // Launch non-coalesced kernel
    copyDataNonCoalesced<<<numBlocks, blockSize>>>(in, out, n);
    hipDeviceSynchronize();

    initializeArray(out, n); // Reset output array

    // Launch coalesced kernel
    copyDataCoalesced<<<numBlocks, blockSize>>>(in, out, n);
    hipDeviceSynchronize();

    hipFree(in);
    hipFree(out);

    return 0;
}
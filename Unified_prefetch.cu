#include "hip/hip_runtime.h"
// //todo: unified memory prefetch
//! unified memory la async ?

#include <stdio.h>
#include <cassert>
#include <iostream>

using std::cout;


__global__ void vectorAdd(int *a, int *b, int *c, int N)
{
    
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;

    if (tid < N)
    {
        c[tid] = a[tid] + b[tid];
    }
}

int main()
{
    const int N = 1 << 16;
    size_t bytes = N * sizeof(int);

    int *a, *b, *c;

    hipMallocManaged(&a, bytes);
    hipMallocManaged(&b, bytes);
    hipMallocManaged(&c, bytes);

    // Get the device ID for prefetching calls
    int id = hipGetDevice(&id);

    // Set some hints about the data and do some prefetching
    hipMemAdvise(a, bytes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
    hipMemAdvise(b, bytes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
    hipMemPrefetchAsync(c, bytes, id);

    // Initialize vectors
    for (int i = 0; i < N; i++)
    {
        a[i] = rand() % 100;
        b[i] = rand() % 100;
    }

    // Pre-fetch 'a' and 'b' arrays to the specified device (GPU)
    hipMemAdvise(a, bytes, hipMemAdviseSetReadMostly, id);
    hipMemAdvise(b, bytes, hipMemAdviseSetReadMostly, id);
    hipMemPrefetchAsync(a, bytes, id);
    hipMemPrefetchAsync(b, bytes, id);


    int BLOCK_SIZE = 1 << 10;

    int GRID_SIZE = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    vectorAdd<<<GRID_SIZE, BLOCK_SIZE>>>(a, b, c, N);

  
    hipDeviceSynchronize();
    

    // Prefetch to the host (CPU)
    hipMemPrefetchAsync(a, bytes, hipCpuDeviceId);
    hipMemPrefetchAsync(b, bytes, hipCpuDeviceId);
    hipMemPrefetchAsync(c, bytes, hipCpuDeviceId);

    // Verify the result on the CPU
    for (int i = 0; i < N; i++)
    {
        assert(c[i] == a[i] + b[i]);
    }

    // Free unified memory (same as memory allocated with hipMalloc)
    hipFree(a);
    hipFree(b);
    hipFree(c);

    cout << "COMPLETED SUCCESSFULLY!\n";

    return 0;
}